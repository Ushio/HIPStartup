#include "hip/hip_runtime.h"
#include "typedbuffer.hpp"
#include "hip/hip_vector_types.h"

__device__
int sqr(int x)
{
	return x * x;
}

extern "C" __global__ void kernelMain( TypedBuffer<int> xs, int value )
{
	int tid = threadIdx.x + blockDim.x * blockIdx.x;
	xs[tid] = sqr(tid) + value;
}